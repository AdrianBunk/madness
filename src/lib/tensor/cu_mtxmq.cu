/*
  This file is part of MADNESS.

  Copyright (C) 2007,2010 Oak Ridge National Laboratory

  This program is free software; you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation; either version 2 of the License, or
  (at your option) any later version.

  This program is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program; if not, write to the Free Software
  Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA

  For more information please contact:

  Robert J. Harrison
  Oak Ridge National Laboratory
  One Bethel Valley Road
  P.O. Box 2008, MS-6367

  email: harrisonrj@ornl.gov
  tel:   865-241-3937
  fax:   865-572-0680

  $Id$
  
  Part of the code is adopted from Nvidia CUDA sample code and NOT OPTMIZED
*/
#ifndef MADNESS_TENSOR_CU_MTXMQ_H__INCLUDED
#define MADNESS_TENSOR_CU_MTXMQ_H__INCLUDED


#include <madness_config.h>
#define ENABLE_CUBLAS 1  
#include <tensor/cu_mtxmq_kernels.cu>
#include <tensor/cu_mtxmq.h>
//#include <world/cuda_streams.h>
#include <hipblas.h>
//namespace madness {

 

    /// Matrix = Matrix transpose * matrix ... reference implementation

    /// Does \c C=AT*B whereas mTxm does C=C+AT*B.  It also supposed
    /// to be fast which it achieves thru restrictions
    ///   * All dimensions even
    ///   * All pointers aligned
    /// \code
    ///    c(i,j) = sum(k) a(k,i)*b(k,j)  <------ does not accumulate into C
    /// \endcode
/*
 template <typename T, typename T, typename tensorT>
 void padwrapper(long dimi, long dimj, const double* pc, T* t0, T* t1, tensorT d,unsigned int i){

	tensorT t = d;
	long nij = dimi*dimj;
	if (IS_ODD(dimi) || IS_ODD(dimj) ||
		IS_UNALIGNED(pc) || IS_UNALIGNED(t0) || IS_UNALIGNED(t1)) {
	    for (long i=0; i<nij; ++i) t0[i] = 0.0;
	    mTxm(dimi, dimj, dimj, t0, t.ptr(), pc);
	    for (int n=1; n<t.ndim(); ++n) {
		for (long i=0; i<nij; ++i) t1[i] = 0.0;
		mTxm(dimi, dimj, dimj, t1, t0, pc);
		std::swap(t0,t1);
	    }
	}
	else {
	    //mTxmq(dimi, dimj, dimj, t0, t.ptr(), pc);
	    print("CUDA KERNEL (dim = ",dimi,",",dimj,")\n");
	    cu_mTxmq(dimi, dimj, dimj, t0, t.ptr(), pc, i);
	    for (int n=1; n<t.ndim(); ++n) {
		//mTxmq(dimi, dimj, dimj, t1, t0, pc);
		cu_mTxmq(dimi, dimj, dimj, t1, t0, pc, i);
		std::swap(t0,t1);
	    }
	}

 }
 */
 template <typename aT, typename bT, typename cT>
    void cu_mTxmq(long dimi, long dimj, long dimk,
               cT* restrict c, const aT* a, const bT* b,void *stream) {
        printf("gpu code");
        const aT *h_A= a;
	const bT *h_B= b;
	cT *h_C= c;
	aT *d_A;
	aT *d_odata;
	bT *d_B, *hb;
	cT *d_C, *hc;
	aT *ha; 
	dim3 threads = dim3(BLOCK_SIZE,BLOCK_SIZE);
	dim3 grid = dim3(dimj / threads.x, dimi / threads.y);
	
	//if (dimi%BLOCK_SIZE !=0 || dimj%BLOCK_SIZE!=0){
	dim3 threads_rem, grid_rem;
	  
	//  kernel = &transposeNoBankConflicts; 
	
	//}
	//unsigned int tile_size = sizeof(aT) * TILE_DIM * (TILE_DIM+1);
	//unsigned int tile_sizee = sizeof(aT) * TILE_DIM * (TILE_DIM);
	int size_i = dimi + (BLOCK_SIZE-(dimi%BLOCK_SIZE));
	int size_k = dimk + (BLOCK_SIZE-(dimk%BLOCK_SIZE));
	int size_j = dimj + (BLOCK_SIZE-(dimj%BLOCK_SIZE));
	int i,j;
	if (dimi%BLOCK_SIZE !=0 || dimj%BLOCK_SIZE!=0 || dimk%BLOCK_SIZE!=0){
	
	grid_rem = dim3(size_i / BLOCK_SIZE, size_k / BLOCK_SIZE);
	ha =(aT*) malloc(size_i*size_k*sizeof(aT));
	hb =(bT*) malloc(size_j*size_k*sizeof(bT));
	hc =(cT*) malloc(size_k*size_j*sizeof(cT));
	for ( i=0, j=0;i<dimi*dimk; i++, j++)
	{
	  if (i%dimk==0 && i!=0)
	    j=j+ (BLOCK_SIZE-(dimk%BLOCK_SIZE));
	  ha[j]=h_A[i];
//	   printf("hA[%d]=%f\t ",j,ha[j]);
	}
	
	
	for (  i=0, j=0;i<dimj*dimk; i++, j++)
	{
	  if (i%dimj==0 && j!=0)
	    j=j+ (BLOCK_SIZE-(dimj%BLOCK_SIZE));
	  hb[j]=h_B[i];
	}
	}
	
	int A_size = size_i*size_k*sizeof(aT);
	int B_size = size_k*size_j*sizeof(bT);
	int C_size = size_k*size_j*sizeof(cT);
	
	//printf("A_size=%d\n\n\n",A_size);
	
	hipHostMalloc((void**)&d_A, A_size) ;
	hipHostMalloc((void**)&d_B, B_size) ;
	hipHostMalloc((void**)&d_C, C_size) ;
	hipHostMalloc((void**)&d_odata, A_size) ;
	hipMemcpy(d_A, ha, A_size, hipMemcpyHostToDevice) ; 
	hipMemcpy(d_B, hb, B_size, hipMemcpyHostToDevice) ;
	
	//printf("tile size = %u\n",tile_size);
	if (dimi%BLOCK_SIZE !=0 || dimj%BLOCK_SIZE!=0 || dimk%BLOCK_SIZE!=0){
	transposeNoBankConflicts<aT><<<grid_rem, threads>>>(d_odata,d_A, size_k, size_i, 1);
//if ( hipSuccess != hipGetLastError() )
  //  printf( "Error!\n" );

//	hipDeviceSynchronize();
//	hipMemcpy((void *)ha,(void *) d_odata, A_size, hipMemcpyDeviceToHost) ;
	grid_rem = dim3(size_j / BLOCK_SIZE, size_k / BLOCK_SIZE);
//	  printf("ha[%d]=%f\t ",i,ha[i]);
	matrixMul_coalescing<aT,bT,cT><<< grid_rem, threads >>>(d_C, d_odata, d_B, size_i, size_j);
	
//hipDeviceSynchronize();
	}
	else
	{ transposeNoBankConflicts<aT><<<grid, threads>>>(d_A,d_A, dimk, dimi, 1);
	
	matrixMul_coalescing<aT,bT,cT><<< grid, threads >>>(d_C, d_A, d_B, dimi, dimj);
	}
	//matrixMul_coalescing_rem<aT,bT,cT><<< grid, threads>>>(d_C, d_A, d_B, dimk, dimj, dimi);
	//if (dimi%BLOCK_SIZE !=0 || dimj%BLOCK_SIZE!=0){
	  //threads_rem = dim3(dimj%BLOCK_SIZE,dimi%BLOCK_SIZE);
	//dim3 grid_rem = dim3(1);
	//matrixMul_coalescing_rem<<< grid_rem, threads_rem, tile_sizee >>>(d_C, d_A, d_B, dimk, dimj, dimi);
	//}
	// copy result from device to host
	hipMemcpy((void *)hc,(void *) d_C, C_size, hipMemcpyDeviceToHost) ;
       for (  i=0, j=0;i<dimj*dimk; i++, j++)
	{
	  if (i%dimj==0 && i!=0)
	    j=j+ (BLOCK_SIZE-(dimj%BLOCK_SIZE));
	  h_C[i]=hc[j];
	}

	free(ha);
	free(hb);
	free(hc);
	hipFree(d_odata);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
    }

template <> void cu_mTxmq(long m, long n,long k, std::complex<double> *C, const std::complex<double> *A, const double *B,void *stream){}    
#if !ENABLE_CUBLAS 
      template void cu_mTxmq(long dimi, long dimj, long dimk, float*  c, const float* a, const float* b,void *stream) ;
     
      template void cu_mTxmq(long m, long n,long k, double *C, const double *A, const double *B,void *stream);
    
  template <> void cu_mTxmq(long m, long n,long k, std::complex<double> *C, const std::complex<double> *A, const std::complex<double> *B,void *stream){}
	 
       
#else

  template<>   void cu_mTxmq(long m, long n,long k, double *C, const double *A, const double *B,void *GPU_stream){


	double one=1.0;
	double zero=0.0;
	printf(" GPU Scublas code execution");
	//sleep(100);
	int M = (int)m;
	int N = (int)n;
	int K = (int)k;
	hipblasStatus_t statt;
	hipError_t stat;	
	double *devPtrA, *devPtrB, *devPtrC;
        hipblasHandle_t handle;	
	hipblasCreate(&handle);
	hipStream_t *stream=(hipStream_t*)GPU_stream;
	hipblasSetStream(handle, *stream);
	
	stat = hipHostMalloc ( (void**)&devPtrA,M*K*sizeof(double),hipHostMallocDefault ) ;
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	stat = hipHostMalloc ((void**)&devPtrB,K*N*sizeof(double),hipHostMallocDefault ) ;
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	stat = hipHostMalloc ((void**)&devPtrC,M*N*sizeof(double),hipHostMallocDefault ) ;
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	hipblasSetMatrixAsync (M, K, sizeof(double), (void *)A, M, (void *)devPtrA, M,*stream);
	hipblasSetMatrixAsync (K, N, sizeof(double), (void *)B, K, (void *)devPtrB, K,*stream);
	//dgemm_("n","t",&nj,&ni,&nk,&one,b,&nj,a,&ni,&zero,c,&nj,1,1);
	//hipblasDgemm('t','n',M,N,K,one,devPtrA,K,devPtrB,K,zero,devPtrC,M);
	hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,N,M,K,&one,devPtrB,N,devPtrA,M,&zero,devPtrC,N);
	int  b=cublasGetError();
	if (b == HIPBLAS_STATUS_INVALID_VALUE)
	  printf("HIPBLAS_STATUS_INVALID_VALUE");
	else if (b == HIPBLAS_STATUS_ARCH_MISMATCH)
	  printf("HIPBLAS_STATUS_ARCH_MISMATCH");
        else if (b ==HIPBLAS_STATUS_EXECUTION_FAILED )
          printf("HIPBLAS_STATUS_EXECUTION_FAILED");
        else if (b ==HIPBLAS_STATUS_MAPPING_ERROR )
          printf("HIPBLAS_STATUS_MAPPING_ERROR");
        else if (b ==HIPBLAS_STATUS_ALLOC_FAILED )
          printf("HIPBLAS_STATUS_ALLOC_FAILED");
        else if (b ==HIPBLAS_STATUS_NOT_INITIALIZED )
          printf("init HIPBLAS_STATUS_NOT_INITIALIZED");
        else if (b ==HIPBLAS_STATUS_INTERNAL_ERROR )
          printf("HIPBLAS_STATUS_INTERNAL_ERROR");
	//else
	  //printf("Error=%d",b);
	//hipblasGetMatrix (K, K, sizeof(double), (void *)devPtrC, K, (void *)C, K);
	//dgemm_("n","t",&nj,&ni,&nk,&one,b,&nj,a,&ni,&zero,c,&nj,1,1);
	//hipblasSgemm('n','t',N,M,K,one,devPtrB,N,devPtrA,M,zero,devPtrC,N);
	hipblasGetMatrixAsync (M, N, sizeof(double), (void *)devPtrC, M, (void *)C, M,*stream);
	hipFree (devPtrA);
	hipFree (devPtrB);
	hipFree (devPtrC);
	hipblasDestroy(handle);

    }


/*	
//hipSetDevice(3);
	double one=1.0;
	double zero=0.0;
	printf(" GPU cublas code execution m=%d, n%d,k=%d",m,n,k);
	//sleep(100);
	int M = (int)m;
	int N = (int)n;
	int K = (int)k;
	hipblasStatus_t stat,b;
	double *devPtrA, *devPtrB, *devPtrC;
 		
	do{
	stat=hipblasCreate(hipblasHandle_t *handle);
	}while(	stat!= hipSuccess);
    b=cublasGetError();
        if (b == HIPBLAS_STATUS_INVALID_VALUE)
          printf("HIPBLAS_STATUS_INVALID_VALUE");
        else if (b == HIPBLAS_STATUS_ARCH_MISMATCH)
          printf("HIPBLAS_STATUS_ARCH_MISMATCH");
else if (b ==HIPBLAS_STATUS_EXECUTION_FAILED )
          printf("HIPBLAS_STATUS_EXECUTION_FAILED");
else if (b ==HIPBLAS_STATUS_MAPPING_ERROR )
          printf("HIPBLAS_STATUS_MAPPING_ERROR");
else if (b ==HIPBLAS_STATUS_ALLOC_FAILED )
          printf("HIPBLAS_STATUS_ALLOC_FAILED");
else if (b ==HIPBLAS_STATUS_NOT_INITIALIZED )
          printf("init HIPBLAS_STATUS_NOT_INITIALIZED");
else if (b ==HIPBLAS_STATUS_INTERNAL_ERROR )
          printf("HIPBLAS_STATUS_INTERNAL_ERROR");

        else
          printf("hipSuccess");
	
	stat = cublasAlloc (M*K, sizeof(double), (void**)&devPtrA);
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	stat = cublasAlloc (K*N, sizeof(double), (void**)&devPtrB);
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	stat = cublasAlloc (M*N*4, sizeof(double), (void**)&devPtrC);
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	hipblasSetMatrixAsync (M, K, sizeof(double), (void *)A, M, (void *)devPtrA, M,stream[i]);
	hipblasSetMatrixAsync (K, N, sizeof(double), (void *)B, K, (void *)devPtrB, K,stream[i]);

//do{	
//	cublasFree (devPtrC);
//	stat = cublasAlloc (M*N, sizeof(double), (void**)&devPtrC);
  //      if (stat != hipSuccess) {
    //    printf ("device memory allocation failed");
      //  return ;
       // }

	hipblasDgemm('t','n',M,N,K,one,devPtrA,M,devPtrB,K,zero,devPtrC,K);
//hipDeviceSynchronize();

	//hipblasDgemm('n','t',N,M,K,one,devPtrB,N,devPtrA,M,zero,devPtrC,N);
	 b=cublasGetError();
//}while(b != hipSuccess);
	if (b == HIPBLAS_STATUS_INVALID_VALUE)
	  printf("HIPBLAS_STATUS_INVALID_VALUE");
	else if (b == HIPBLAS_STATUS_ARCH_MISMATCH)
	  printf("HIPBLAS_STATUS_ARCH_MISMATCH");
else if (b ==HIPBLAS_STATUS_EXECUTION_FAILED )
          printf("HIPBLAS_STATUS_EXECUTION_FAILED");
else if (b ==HIPBLAS_STATUS_MAPPING_ERROR )
          printf("HIPBLAS_STATUS_MAPPING_ERROR");
else if (b ==HIPBLAS_STATUS_ALLOC_FAILED )
          printf("HIPBLAS_STATUS_ALLOC_FAILED");
else if (b ==HIPBLAS_STATUS_NOT_INITIALIZED )
          printf("HIPBLAS_STATUS_NOT_INITIALIZED");
else if (b ==HIPBLAS_STATUS_INTERNAL_ERROR )
          printf("HIPBLAS_STATUS_INTERNAL_ERROR");

	else
	  printf("hipSuccess");

// make sure Dgemm is finished
        hipError_t cudaErr = hipDeviceSynchronize();
        if( cudaErr != hipSuccess ) {
    printf( "Dgemm failed on invocation \n" );
        }
	//hipblasGetMatrix (K, K, sizeof(double), (void *)devPtrC, K, (void *)C, K);
	//dgemm_("n","t",&nj,&ni,&nk,&one,b,&nj,a,&ni,&zero,c,&nj,1,1);
	//hipblasDgemm('n','t',N,M,K,one,devPtrB,N,devPtrA,M,zero,devPtrC,N);
	hipblasGetMatrixAsync (M, N, sizeof(double), (void *)devPtrC, M, (void *)C, M,stream[i]);
	cublasFree (devPtrA);
	cublasFree (devPtrB);
	cublasFree (devPtrC);
	hipblasDestroy(hipblasHandle_t handle);
    }
    
  */  
  template<>   void cu_mTxmq(long m, long n,long k, std::complex<double> *C, const std::complex<double> *A, const std::complex<double> *B,void *GPU_stream){
	hipDoubleComplex one;
	one.x=1.0;
	one.y=0.0;
	hipDoubleComplex zero;
	zero.x=0.0;
	zero.y=0.0;
	printf(" complx GPU code execution");
	//sleep(100);
	int M = (int)m;
	int N = (int)n;
	int K = (int)k;
	
	hipError_t  stat;
	hipDoubleComplex *devPtrA, *devPtrB, *devPtrC;
	hipDoubleComplex *A1=(hipDoubleComplex *)A;
	hipDoubleComplex *B1=(hipDoubleComplex *)B;
	//hipDoubleComplex *C1=(hipDoubleComplex *)C;
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipStream_t *stream=(hipStream_t*)GPU_stream;
	hipblasSetStream(handle, *stream);
	stat = hipHostMalloc ( (void**)&devPtrA,M*K*sizeof(double),hipHostMallocDefault ) ;
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	stat = hipHostMalloc ((void**)&devPtrB,K*N*sizeof(double),hipHostMallocDefault ) ;
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	stat = hipHostMalloc ((void**)&devPtrC,M*N*sizeof(double),hipHostMallocDefault ) ;
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	hipblasSetMatrixAsync (M, K, sizeof(hipDoubleComplex), (void *)A1, M, (void *)devPtrA, M, *stream);
	hipblasSetMatrixAsync (K, N, sizeof(hipDoubleComplex), (void *)B1, K, (void *)devPtrB, K, *stream);
	
	//hipblasZgemm('n','t',N,M,K,one,devPtrB,N,devPtrA,M,zero,devPtrC,K);
	hipblasZgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,N,M,K,&one,devPtrB,N,devPtrA,M,&zero,devPtrC,N);
	hipblasGetMatrixAsync (N, M, sizeof(*C), (void *)devPtrC, N, (void *)C, N, *stream);
	hipFree (devPtrA);
	hipFree (devPtrB);
	hipFree (devPtrC);
	hipblasDestroy( handle);

    }


template<>  void cu_mTxmq(long m, long n,long k,float *C, const float *A, const float *B,void *GPU_stream){
	float one=1.0;
	float zero=0.0;
	printf(" GPU Scublas code execution");
	//sleep(100);
	int M = (int)m;
	int N = (int)n;
	int K = (int)k;
	hipError_t stat;
	float *devPtrA, *devPtrB, *devPtrC;
	hipblasHandle_t handle;
        hipblasCreate(&handle);
        hipStream_t *stream=(hipStream_t*)GPU_stream;
        hipblasSetStream(handle, *stream);	
	stat = hipHostMalloc ( (void**)&devPtrA,M*K*sizeof(double),hipHostMallocDefault ) ;
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	stat = hipHostMalloc ((void**)&devPtrB,K*N*sizeof(double),hipHostMallocDefault ) ;
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	stat = hipHostMalloc ((void**)&devPtrC,M*N*sizeof(double),hipHostMallocDefault ) ;
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	hipblasSetMatrixAsync (M, K, sizeof(float), (void *)A, M, (void *)devPtrA, M,*stream);
	hipblasSetMatrixAsync (K, N, sizeof(float), (void *)B, K, (void *)devPtrB, K,*stream);
	//dgemm_("n","t",&nj,&ni,&nk,&one,b,&nj,a,&ni,&zero,c,&nj,1,1);
	//hipblasDgemm('t','n',M,N,K,one,devPtrA,K,devPtrB,K,zero,devPtrC,M);
	hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,N,M,K,&one,devPtrB,N,devPtrA,M,&zero,devPtrC,N);
	int  b=cublasGetError();
	if (b == HIPBLAS_STATUS_INVALID_VALUE)
	  printf("HIPBLAS_STATUS_INVALID_VALUE");
	else if (b == HIPBLAS_STATUS_ARCH_MISMATCH)
	  printf("HIPBLAS_STATUS_ARCH_MISMATCH");
	else
	  printf("Error=%d",b);
	//hipblasGetMatrix (K, K, sizeof(float), (void *)devPtrC, K, (void *)C, K);
	//dgemm_("n","t",&nj,&ni,&nk,&one,b,&nj,a,&ni,&zero,c,&nj,1,1);
	//hipblasSgemm('n','t',N,M,K,one,devPtrB,N,devPtrA,M,zero,devPtrC,N);
	hipblasGetMatrixAsync (M, N, sizeof(float), (void *)devPtrC, M, (void *)C, M,*stream);
	hipFree (devPtrA);
	hipFree (devPtrB);
	hipFree (devPtrC);
	hipblasDestroy(handle);

    }
    
    
  template<>   void cu_mTxmq(long m, long n,long k, std::complex<float> *C, const std::complex<float> *A, const std::complex<float> *B,void *GPU_stream){
	hipComplex one;
	one.x=1.0;
	one.y=0.0;
	hipComplex zero;
	zero.x=0.0;
	zero.y=0.0;
	printf(" complx GPU code execution");
	//sleep(100);
	int M = (int)m;
	int N = (int)n;
	int K = (int)k;
	
	hipError_t  stat;
	hipComplex *devPtrA, *devPtrB, *devPtrC;
	hipComplex *A1=(hipComplex *)A;
	hipComplex *B1=(hipComplex *)B;
	//hipDoubleComplex *C1=(hipDoubleComplex *)C;
	hipblasHandle_t handle;
        hipblasCreate(&handle);
        hipStream_t *stream=(hipStream_t*)GPU_stream;
        hipblasSetStream(handle, *stream);
	stat = hipHostMalloc ( (void**)&devPtrA,M*K*sizeof(double),hipHostMallocDefault ) ;
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	stat = hipHostMalloc ((void**)&devPtrB,K*N*sizeof(double),hipHostMallocDefault ) ;
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	
	stat = hipHostMalloc ((void**)&devPtrC,M*N*sizeof(double),hipHostMallocDefault ) ;
	if (stat != hipSuccess) {
	printf ("device memory allocation failed");
	return ;
	}
	hipblasSetMatrixAsync (M, K, sizeof(hipComplex), (void *)A1, M, (void *)devPtrA, M,*stream);
	hipblasSetMatrixAsync (K, N, sizeof(hipComplex), (void *)B1, K, (void *)devPtrB, K,*stream);
	
	//hipblasCgemm('n','t',N,M,K,one,devPtrB,N,devPtrA,M,zero,devPtrC,K);
	hipblasCgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_T,N,M,K,&one,devPtrB,N,devPtrA,M,&zero,devPtrC,N);
	hipblasGetMatrixAsync (N, M, sizeof(*C), (void *)devPtrC, N, (void *)C, N,*stream);
	hipFree (devPtrA);
	hipFree (devPtrB);
	hipFree (devPtrC);
	hipblasDestroy( handle);

    }
#endif

//}
#endif // MADNESS_TENSOR_CU_MTXMQ_H__INCLUDED


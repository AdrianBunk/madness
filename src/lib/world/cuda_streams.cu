#include <hip/hip_runtime.h>
//#include <world/cuda_streams.h>
#include <stdio.h>

#include <hipblas.h>
//#define NUM_STREAMS 16
//cudaStream_t streams[NUM_STREAMS];

extern "C" void * cublashandle_create(){
    hipblasHandle_t * handle = new hipblasHandle_t;
    hipblasCreate(handle);
    void * h = (void *)handle;
    return h;
}

extern "C" void cublashandle_destroy(void * h){
    hipblasHandle_t * handle = (hipblasHandle_t*)h;
    hipblasDestroy(*handle);
    delete handle;
}

extern "C" void ** streams_initialize(unsigned int streams, void * h){
    unsigned int i;
    hipblasHandle_t * handle = (hipblasHandle_t *)h;
    void ** cast_streams = new void*[streams];
    for (i = 0; i < streams; i++){
        hipStream_t * stream = new hipStream_t;
        hipStreamCreate(stream);
        /*
        cudaStream_t * gs;
        cudaError_t err = cudaMalloc((void **)&gs, sizeof(cudaStream_t));
        if (err != cudaSuccess){
          printf("cudaMalloc fail");
          exit(-1);
        }
        err = cudaMemcpy(gs, stream, sizeof(cudaStream_t), cudaMemcpyHostToDevice); 
        if (err != cudaSuccess){
          printf("cudaMempcy fail");
          exit(-1);
        }
        cudaStreamCreate(gs);*/
        //cublasSetStream(*handle, *stream);
        //cublasSetStream(*handle, *gs);
        cast_streams[i] = (void *)stream;
        //cast_streams[i] = (void *)gs;
    }
    return cast_streams;
}


extern "C" void streams_destroy(void ** cast_streams, unsigned int streams){
    unsigned int i;
    for (i = 0; i < streams; i++){
        hipStream_t * stream = (hipStream_t *)cast_streams[i];
        hipStreamDestroy(*stream);
        delete stream;
    }
}

extern "C" void streams_synchronize(void ** cast_streams, unsigned int streams){
    unsigned int i;
    for (i = 0; i < streams; i++){
        hipStream_t * stream = (hipStream_t *)cast_streams[i];
        hipStreamSynchronize(*stream);
    }

}

#include <hip/hip_runtime.h>
//#include <world/cuda_streams.h>

//#define NUM_STREAMS 16
//cudaStream_t streams[NUM_STREAMS];

extern "C" void ** streams_initialize(unsigned int streams){
    unsigned int i;
    void ** cast_streams = new void*[streams];
    for (i = 0; i < streams; i++){
        hipStream_t * stream = new hipStream_t;
        hipStreamCreate(stream);
        cast_streams[i] = (void *)stream;
    }
    return cast_streams;
}


extern "C" void streams_destroy(void ** cast_streams, unsigned int streams){
    unsigned int i;
    for (i = 0; i < streams; i++){
        hipStream_t * stream = (hipStream_t *)cast_streams[i];
        hipStreamDestroy(*stream);
        delete stream;
    }
}

extern "C" void streams_synchronize(void ** cast_streams, unsigned int streams){
    unsigned int i;
    for (i = 0; i < streams; i++){
        hipStream_t * stream = (hipStream_t *)cast_streams[i];
        hipStreamSynchronize(*stream);
    }

}
